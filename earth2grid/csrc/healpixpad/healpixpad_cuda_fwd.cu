#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Written by Mauro Bisson <maurob@nvidia.com> and Thorsten Kurth <tkurth@nvidia.com>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <torch/extension.h>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/CUDAUtils.h>
#include "cudamacro.h"
#include "healpixpad.h"

#define THREADS  64

#define DIV_UP(a,b) (((a)+((b)-1))/(b))

// All coordinates are w.r.t. a face[dimK][dimL][dimM]:
//
//     ^ k-axis
//    /
//   *---------*
//  /.        /|
// *---------*-+---> m-axis
// | .       | |
// | .       | |
// | *.......|.*
// |.        |/
// *---------*
// |
// |
// \/ l-axis
//
// Along the k-axis, dimJ=12 faces form a "sphere" and
// we have in total dimI sphere in the buffers

template<typename VAL_T, bool CHANNELS_LAST>
__global__ void HEALPixPadFwd_bulk_vec_k(
        const int padSize,
        const int dimI, const int dimJ,
        const int dimK, const int dimL, const int dimM,
        torch::PackedTensorAccessor32<VAL_T,5,torch::RestrictPtrTraits> vin,
        torch::PackedTensorAccessor32<VAL_T,5,torch::RestrictPtrTraits> vout)
{
  using VecT = typename VecTraits<VAL_T>::VecT;
  constexpr int W = VecTraits<VAL_T>::LANE_WIDTH;

  const long long tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= ((long long)dimI)*dimJ*dimK*dimL*dimM / W) {
    return;
  }

  const int dimKVec = CHANNELS_LAST ? dimK / W : dimK;
  const int dimMVec = CHANNELS_LAST ? dimM : dimM / W;

  int i,j,k,l,m;
  if constexpr(CHANNELS_LAST) {
    k = (tid % dimKVec) * W;
    m = (tid / dimKVec) % dimM;
    l = (tid / (dimKVec * dimM)) % dimL;
    j = (tid / (dimKVec * dimM * dimL)) % dimJ;
    i =  tid / (dimKVec * dimM * dimL * dimJ);
  } else {
    m = (tid % dimMVec) * W;
    l = (tid / dimMVec) % dimL;
    k = (tid / (dimMVec * dimL)) % dimK;
    j = (tid / (dimMVec * dimL * dimK)) % dimJ;
    i =  tid / (dimMVec * dimL * dimK * dimJ);
  }

  const VecT srcVec = *reinterpret_cast<const VecT*>(
              &getElem<VAL_T,CHANNELS_LAST>(vin,i,j,k,l,m));

  if (!CHANNELS_LAST && ((padSize & (W - 1)) != 0)) {
    // Store unvectorized as padding makes output unaligned
    const VAL_T* lanes = reinterpret_cast<const VAL_T*>(&srcVec);
#pragma unroll
    for (int w = 0; w < W; ++w) {
        getElemMutable<VAL_T,CHANNELS_LAST>(
            vout, i, j, k, padSize + l, padSize + m + w) = lanes[w];
    }
  } else {
    // src and dst are both aligned
    VecT* dstVec = reinterpret_cast<VecT*>(
      &getElemMutable<VAL_T,CHANNELS_LAST>(vout,i,j,k, padSize+l, padSize+m));
    *dstVec = srcVec;
  }
}

template<typename VAL_T, bool CHANNELS_LAST>
__global__ void HEALPixPadFwd_bulk_k(const int padSize,
				     const int dimI,
				     const int dimJ,
				     const int dimK,
				     const int dimL,
				     const int dimM,
				     const torch::PackedTensorAccessor32<VAL_T, 5, torch::RestrictPtrTraits> vin,
				     torch::PackedTensorAccessor32<VAL_T, 5, torch::RestrictPtrTraits> vout) {

  const long long tid = ((long long)blockIdx.x)*blockDim.x + threadIdx.x;

  if (tid >= ((long long)dimI)*dimJ*dimK*dimL*dimM) {
    return;
  }

  // compute individual indices
  int i,j,k,l,m;
  if constexpr(CHANNELS_LAST) {
    k = tid % dimK;
    m = (tid / dimK) % dimM;
    l = (tid / (dimK * dimM)) % dimL;
    j = (tid / (dimK * dimM * dimL)) % dimJ;
    i = tid / (dimK * dimM * dimL * dimJ);
  } else {
    m = (tid % (dimM*dimL)) % dimM;
    l = (tid % (dimM*dimL)) / dimM;
    k = (tid % (dimM*dimL*dimK)) / (dimM*dimL);
    j = (tid % (dimM*dimL*dimK*dimJ)) / (dimM*dimL*dimK);
    i = (tid / (dimJ * dimK * dimL * dimM));
  }

  // copy data
  getElemMutable<VAL_T, CHANNELS_LAST>(vout, i, j, k, padSize+l, padSize+m) = getElem<VAL_T, CHANNELS_LAST>(vin, i, j, k, l, m);

  return;
}

template<typename VAL_T, bool CHANNELS_LAST>
__device__ VAL_T getT_d(const int k,
			const int p,
			const int m,
			const int dimL,
			const int dimM,
			const int faceId,
			const int sphrId,
			const torch::PackedTensorAccessor32<VAL_T, 5, torch::RestrictPtrTraits> sphr) {

  VAL_T ret = VAL_T(0);

  switch(faceId) {
      // north faces
    case  0: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 1, k, m, p); break;
    case  1: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 2, k, m, p); break;
    case  2: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 3, k, m, p); break;
    case  3: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 0, k, m, p); break;
      // center faces
    case  4: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 0, k, dimL-1-p, m); break;
    case  5: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 1, k, dimL-1-p, m); break;
    case  6: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 2, k, dimL-1-p, m); break;
    case  7: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 3, k, dimL-1-p, m); break;
      // south faces
    case  8: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 5, k, dimL-1-p, m); break;
    case  9: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 6, k, dimL-1-p, m); break;
    case 10: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 7, k, dimL-1-p, m); break;
    case 11: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 4, k, dimL-1-p, m); break;
    }

  return ret;
}

template<typename VAL_T, bool CHANNELS_LAST>
__device__ VAL_T getB_d(const int k,
			const int p,
			const int m,
			const int dimL,
			const int dimM,
			const int faceId,
			const int sphrId,
			const torch::PackedTensorAccessor32<VAL_T, 5, torch::RestrictPtrTraits> sphr) {

  VAL_T ret = VAL_T(0);

  switch(faceId) {
      // north faces
    case  0: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  4, k, p, m); break;
    case  1: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  5, k, p, m); break;
    case  2: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  6, k, p, m); break;
    case  3: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  7, k, p, m); break;
      // center faces
    case  4: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 11, k, p, m); break;
    case  5: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  8, k, p, m); break;
    case  6: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  9, k, p, m); break;
    case  7: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 10, k, p, m); break;
      // south faces
    case  8: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 11, k, m, dimM-1-p); break;
    case  9: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  8, k, m, dimM-1-p); break;
    case 10: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  9, k, m, dimM-1-p); break;
    case 11: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 10, k, m, dimM-1-p); break;
    }

  return ret;
}

template<typename VAL_T, bool CHANNELS_LAST>
__device__ VAL_T getL_d(const int k,
			const int p,
			const int m,
			const int dimL,
			const int dimM,
			const int faceId,
			const int sphrId,
			const torch::PackedTensorAccessor32<VAL_T, 5, torch::RestrictPtrTraits> sphr) {

  VAL_T ret = VAL_T(0);

    switch(faceId) {
      // north faces
    case  0: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 3, k, p, m); break;
    case  1: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 0, k, p, m); break;
    case  2: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 1, k, p, m); break;
    case  3: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 2, k, p, m); break;
      // center faces
    case  4: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 3, k, m, dimM-1-p); break;
    case  5: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 0, k, m, dimM-1-p); break;
    case  6: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 1, k, m, dimM-1-p); break;
    case  7: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 2, k, m, dimM-1-p); break;
      // south faces
    case  8: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 4, k, m, dimM-1-p); break;
    case  9: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 5, k, m, dimM-1-p); break;
    case 10: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 6, k, m, dimM-1-p); break;
    case 11: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 7, k, m, dimM-1-p); break;
    }

  return ret;
}

  template<typename VAL_T, bool CHANNELS_LAST>
__device__ VAL_T getR_d(const int k,
			const int p,
			const int m,
			const int dimL,
			const int dimM,
			const int faceId,
			const int sphrId,
      const torch::PackedTensorAccessor32<VAL_T, 5, torch::RestrictPtrTraits> sphr) {

  VAL_T ret = VAL_T(0);

    switch(faceId) {
      // north faces
    case  0: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  5, k, m, p); break;
    case  1: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  6, k, m, p); break;
    case  2: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  7, k, m, p); break;
    case  3: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  4, k, m, p); break;
      // center faces
    case  4: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  8, k, m, p); break;
    case  5: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  9, k, m, p); break;
    case  6: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 10, k, m, p); break;
    case  7: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 11, k, m, p); break;
      // south faces
    case  8: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  9, k, dimL-1-p, m); break;
    case  9: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 10, k, dimL-1-p, m); break;
    case 10: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 11, k, dimL-1-p, m); break;
    case 11: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  8, k, dimL-1-p, m); break;
    }

  return ret;
}

template<typename VAL_T, bool CHANNELS_LAST>
__device__ VAL_T getTL_d(const int padSize,
			 const int p,
			 const int q,
			 const int k,
			 const int dimL,
			 const int dimM,
			 const int faceId,
			 const int sphrId,
			 const torch::PackedTensorAccessor32<VAL_T, 5, torch::RestrictPtrTraits> sphr) {

  VAL_T ret = VAL_T(0);

  const int pinv = padSize-1 - p;
  const int qinv = padSize-1 - q;

  switch(faceId) {
    // north faces
  case  0: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 2, k, pinv, qinv); break;
  case  1: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 3, k, pinv, qinv); break;
  case  2: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 0, k, pinv, qinv); break;
  case  3: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 1, k, pinv, qinv); break;
    // center faces
  case  4:
  case  5:
  case  6:
  case  7: {
    int srcTRface;
    int srcBLface;
    switch(faceId) {
    case  4: srcTRface = 3; srcBLface = 0; break;
    case  5: srcTRface = 0; srcBLface = 1; break;
    case  6: srcTRface = 1; srcBLface = 2; break;
    case  7: srcTRface = 2; srcBLface = 3; break;
    }
    if (p == q)  {
      ret = (getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, srcTRface, k, 0, dimM-1-qinv) \
              + getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, srcBLface, k, dimL-1-pinv, 0)) / VAL_T(2);
      break;
    } else if (p > q)  {
      ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, srcTRface, k, p-1-q, dimM-1-qinv);
      break;
    } else  /* p < q*/ {
      ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, srcBLface, k, dimL-1-pinv, q-1-p);
      break;
    }
  }
    // south faces
  case  8: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 0, k, dimL-pinv, -qinv-1); break;
  case  9: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 1, k, dimL-pinv, -qinv-1); break;
  case 10: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 2, k, dimL-pinv, -qinv-1); break;
  case 11: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 3, k, dimL-pinv, -qinv-1); break;
  }
  return ret;
}

template<typename VAL_T, bool CHANNELS_LAST>
__device__ VAL_T getTR_d(const int padSize,
			 const int p,
			 const int q,
			 const int k,
			 const int dimL,
			 const int dimM,
			 const int faceId,
			 const int sphrId,
       const torch::PackedTensorAccessor32<VAL_T, 5, torch::RestrictPtrTraits> sphr) {

  VAL_T ret = VAL_T(0);

  const int pinv = padSize-1 - p;

  switch(faceId) {
    // north faces
  case  0: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  1, k, dimL-1-pinv, q); break;
  case  1: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  2, k, dimL-1-pinv, q); break;
  case  2: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  3, k, dimL-1-pinv, q); break;
  case  3: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  0, k, dimL-1-pinv, q); break;
    // center faces
  case  4: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  5, k, dimL-1-pinv, q); break;
  case  5: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  6, k, dimL-1-pinv, q); break;
  case  6: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  7, k, dimL-1-pinv, q); break;
  case  7: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  4, k, dimL-1-pinv, q); break;
    // south faces
  case  8: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  9, k, dimL-1-pinv, q); break;
  case  9: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 10, k, dimL-1-pinv, q); break;
  case 10: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 11, k, dimL-1-pinv, q); break;
  case 11: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  8, k, dimL-1-pinv, q); break;
  }
  return ret;
}

template<typename VAL_T, bool CHANNELS_LAST>
__device__ VAL_T getBL_d(int padSize,
			 const int p,
			 const int q,
			 const int k,
			 const int dimL,
			 const int dimM,
			 const int faceId,
			 const int sphrId,
       const torch::PackedTensorAccessor32<VAL_T, 5, torch::RestrictPtrTraits> sphr) {

  VAL_T ret = VAL_T(0);

  const int qinv = padSize-1 - q;

  switch(faceId) {
    // north faces
  case  0: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  3, k, p, dimM-1-qinv); break;
  case  1: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  0, k, p, dimM-1-qinv); break;
  case  2: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  1, k, p, dimM-1-qinv); break;
  case  3: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  2, k, p, dimM-1-qinv); break;
    // center faces
  case  4: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  7, k, p, dimM-1-qinv); break;
  case  5: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  4, k, p, dimM-1-qinv); break;
  case  6: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  5, k, p, dimM-1-qinv); break;
  case  7: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  6, k, p, dimM-1-qinv); break;
    // south faces
  case  8: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 11, k, p, dimM-1-qinv); break;
  case  9: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  8, k, p, dimM-1-qinv); break;
  case 10: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  9, k, p, dimM-1-qinv); break;
  case 11: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 10, k, p, dimM-1-qinv); break;
  }
  return ret;
}

template<typename VAL_T, bool CHANNELS_LAST>
__device__ VAL_T getBR_d(const int padSize,
			 const int p,
			 const int q,
			 const int k,
			 const int dimL,
			 const int dimM,
			 const int faceId,
			 const int sphrId,
       const torch::PackedTensorAccessor32<VAL_T, 5, torch::RestrictPtrTraits> sphr) {

  VAL_T ret = VAL_T(0);

  switch(faceId) {
    // north faces
  case  0: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  8, k, p, q); break;
  case  1: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  9, k, p, q); break;
  case  2: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 10, k, p, q); break;
  case  3: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 11, k, p, q); break;
    // center faces
  case  4:
  case  5:
  case  6:
  case  7: {
    int srcTRface;
    int srcBLface;
    switch(faceId) {
    case  4: srcTRface = 11; srcBLface =  8; break;
    case  5: srcTRface =  8; srcBLface =  9; break;
    case  6: srcTRface =  9; srcBLface = 10; break;
    case  7: srcTRface = 10; srcBLface = 11; break;
    }
    if (p == q)  {
      ret = (getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, srcTRface, k, p, dimM-1) \
              + getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, srcBLface, k, dimL-1, q)) / VAL_T(2);
      break;
    } else if (p > q)  {
      ret =  getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, srcTRface, k, p, dimM-(p-q));
      break;
    } else  /* p < q*/ {
      ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, srcBLface, k, dimL-(q-p), q);
      break;
    }
  }
    // south faces
  case  8: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 10, k, dimL-p, -1-q); break;
  case  9: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId, 11, k, dimL-p, -1-q); break;
  case 10: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  8, k, dimL-p, -1-q); break;
  case 11: ret = getElem<VAL_T, CHANNELS_LAST>(sphr, sphrId,  9, k, dimL-p, -1-q); break;
  }
  return ret;
}

template<typename VAL_T, bool CHANNELS_LAST>
__global__ void HEALPixPadFwd_haloSD_k(const int padSize,
				       const int dimI,
				       const int dimJ,
				       const int dimK,
				       const int dimL,
				       const int dimM,
				       const torch::PackedTensorAccessor32<VAL_T, 5, torch::RestrictPtrTraits> input,
				       torch::PackedTensorAccessor32<VAL_T, 5, torch::RestrictPtrTraits> output) {

  const long long tid = ((long long)blockIdx.x)*blockDim.x + threadIdx.x;

  if (tid >= ((long long)dimI)*dimJ*dimK*dimM*padSize) {
    return;
  }

  const long long sphrId = tid / (dimJ*dimK*dimM*padSize);
  const long long faceId = (tid - sphrId*(dimJ*dimK*dimM*padSize)) / (dimK*dimM*padSize);

  const int dimLO = dimL + 2*padSize;
  const int dimMO = dimM + 2*padSize;

  int k, p, m;
  if constexpr (CHANNELS_LAST) {
    k =  tid % dimK;
    p = (tid / dimK)            % padSize;
    m = (tid / (dimK * padSize)) % dimM;
  } else {
    m =  tid % dimM;
    p = (tid / dimM)            % padSize;
    k = (tid / (dimM * padSize)) % dimK;
  }
  // copy top    face
  getElemMutable<VAL_T, CHANNELS_LAST>(output, sphrId, faceId, k, padSize-1-p, padSize+m) = getT_d<VAL_T, CHANNELS_LAST>(k, p, m, dimL, dimM, faceId, sphrId, input);
  // copy bottom face
  getElemMutable<VAL_T, CHANNELS_LAST>(output, sphrId, faceId, k, padSize+dimL+p, padSize+m) = getB_d<VAL_T, CHANNELS_LAST>(k, p, m, dimL, dimM, faceId, sphrId, input);
  // copy left   face
  getElemMutable<VAL_T, CHANNELS_LAST>(output, sphrId, faceId, k, padSize+m, padSize-1-p) = getL_d<VAL_T, CHANNELS_LAST>(k, p, m, dimL, dimM, faceId, sphrId, input);
  // copy right  face
  getElemMutable<VAL_T, CHANNELS_LAST>(output, sphrId, faceId, k, padSize+m, padSize+dimM+p) = getR_d<VAL_T, CHANNELS_LAST>(k, p, m, dimL, dimM, faceId, sphrId, input);

  // padSize is always <= dimM(L)
  // so there are always enough
  // threads to fully cover the
  // corners
  if (m < padSize) {
    const int q = m;
    getElemMutable<VAL_T, CHANNELS_LAST>(output, sphrId, faceId, k, p, q) = getTL_d<VAL_T, CHANNELS_LAST>(padSize, p, q, k, dimL, dimM, faceId, sphrId, input);
    getElemMutable<VAL_T, CHANNELS_LAST>(output, sphrId, faceId, k, p, dimMO-padSize+q) = getTR_d<VAL_T, CHANNELS_LAST>(padSize, p, q, k, dimL, dimM, faceId, sphrId, input);
    getElemMutable<VAL_T, CHANNELS_LAST>(output, sphrId, faceId, k, dimLO-padSize+p, q) = getBL_d<VAL_T, CHANNELS_LAST>(padSize, p, q, k, dimL, dimM, faceId, sphrId, input);
    getElemMutable<VAL_T, CHANNELS_LAST>(output, sphrId, faceId, k, dimLO-padSize+p, dimMO-padSize+q) = getBR_d<VAL_T, CHANNELS_LAST>(padSize, p, q, k, dimL, dimM, faceId, sphrId, input);
  }

  return;
}


template<typename REAL_T, bool CHANNELS_LAST>
void HEALPixPadFwd(int padSize,
		   torch::Tensor input,
		   torch::Tensor output,
		   hipStream_t stream) {

  const int dimI = input.size(0);
  const int dimJ = input.size(1);
  const int dimK = (CHANNELS_LAST ? input.size(4) : input.size(2));
  const int dimL = (CHANNELS_LAST ? input.size(2) : input.size(3));
  const int dimM = (CHANNELS_LAST ? input.size(3) : input.size(4));

  if (dimI*dimJ*dimK*dimL*dimM <= 0) {
    fprintf(stderr, "%s:%d: error, one or more dimension is less than or equal zero!\n", __func__, __LINE__);
    exit(EXIT_FAILURE);
  }

  if (dimJ != 12) {
    fprintf(stderr, "%s:%d: error, dimJ must be equal to 12!\n", __func__, __LINE__);
    exit(EXIT_FAILURE);
  }

  if (dimL != dimM) {
    fprintf(stderr, "%s:%d: error, dimL must be equal to dimM!\n", __func__, __LINE__);
    exit(EXIT_FAILURE);
  }

  if (padSize > dimL) {
    fprintf(stderr, "%s:%d: error, padSize and less than or equal dimL (or dimM)\n", __func__, __LINE__);
    exit(EXIT_FAILURE);
  }

  const int W = VecTraits<REAL_T>::LANE_WIDTH;
  const bool canVec = (((CHANNELS_LAST ? dimK : dimM) & (W-1)) == 0);
  const int nth_b = THREADS;

  if (canVec) {
    const int nbl_b  = DIV_UP(dimI*dimJ*dimK*dimL*dimM/W, nth_b);

    HEALPixPadFwd_bulk_vec_k<REAL_T,CHANNELS_LAST><<<nbl_b, nth_b, 0, stream>>>(padSize, dimI, dimJ, dimK, dimL, dimM,
              input.packed_accessor32<REAL_T,5,torch::RestrictPtrTraits>(),
              output.packed_accessor32<REAL_T,5,torch::RestrictPtrTraits>());

    CHECK_ERROR("HEALPixPadFwd_bulk_vec_k");
  } else {
    const int nbl_b  = DIV_UP(dimI*dimJ*dimK*dimL*dimM, nth_b);

    HEALPixPadFwd_bulk_k<REAL_T,CHANNELS_LAST><<<nbl_b, nth_b, 0, stream>>>(padSize, dimI, dimJ, dimK, dimL, dimM,
              input.packed_accessor32<REAL_T,5,torch::RestrictPtrTraits>(),
              output.packed_accessor32<REAL_T,5,torch::RestrictPtrTraits>());

    CHECK_ERROR("HEALPixPadFwd_bulk_k");
  }

  // copy haloes
  const int nth_f = THREADS;
  const int nbl_f = DIV_UP(dimI*dimJ*dimK*dimM*padSize, nth_f);

  // this also takes care of the corners
  REAL_T* dataIn_d = input.data_ptr<REAL_T>();
  REAL_T* dataOut_d = output.data_ptr<REAL_T>();
  HEALPixPadFwd_haloSD_k<REAL_T, CHANNELS_LAST><<<nbl_f, nth_f, 0, stream>>>(padSize, dimI, dimJ, dimK, dimL, dimM,
						      input.packed_accessor32<REAL_T, 5, torch::RestrictPtrTraits>(),
						      output.packed_accessor32<REAL_T, 5, torch::RestrictPtrTraits>());

  CHECK_ERROR("HEALPixPadFwd_haloTB_k");

  return;
}


std::vector<torch::Tensor> healpixpad_cuda_forward(torch::Tensor input, int pad, bool channels_last) {

  const auto batch_size = input.size(0);
  const auto num_faces = input.size(1);
  const auto num_channels = (channels_last ? input.size(4) : input.size(2));
  const auto face_size = input.size(3);

  // allocate output tensor
  torch::TensorOptions options = torch::TensorOptions().device(input.device()).dtype(input.dtype());
  torch::Tensor output;
  if(!channels_last) {
    output = torch::empty({batch_size, num_faces, num_channels, face_size+2*pad, face_size+2*pad}, options);
  } else {
    output = torch::empty({batch_size, num_faces, face_size+2*pad, face_size+2*pad, num_channels}, options);
  }

  // get cuda stream:
  auto stream = at::cuda::getCurrentCUDAStream().stream();

  // call wrapper
  switch (input.scalar_type()) {
  case torch::ScalarType::Double:
    if (channels_last) HEALPixPadFwd<double, true>(pad, input, output, stream);
    else HEALPixPadFwd<double, false>(pad, input, output, stream);
    break;
  case torch::ScalarType::Float:
    if (channels_last) HEALPixPadFwd<float, true>(pad, input, output, stream);
    else HEALPixPadFwd<float, false>(pad, input, output, stream);
    break;
  case torch::ScalarType::Half:
    if (channels_last) HEALPixPadFwd<at::Half, true>(pad, input, output, stream);
    else  HEALPixPadFwd<at::Half, false>(pad, input, output, stream);
    break;
  case torch::ScalarType::BFloat16:
    if (channels_last) HEALPixPadFwd<at::BFloat16, true>(pad, input, output, stream);
    else HEALPixPadFwd<at::BFloat16, false>(pad, input, output, stream);
    break;
  default:
    throw std::invalid_argument("Unsupported datatype for healpixpad_cuda_forward.");
  }

  return {output};
}
