#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Written by Mauro Bisson <maurob@nvidia.com> and THorsten Kurth <tkurth@nvidia.com>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <torch/extension.h>
#include <c10/cuda/CUDAStream.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cudamacro.h"
#include "healpixpad.h" 

#define THREADS  64

#define MIN(x,y) (((x)<(y))?(x):(y))
#define MAX(x,y) (((x)>(y))?(x):(y))

#define DIV_UP(a,b) (((a)+((b)-1))/(b))

// All coordinates are w.r.t. a face[dimK][dimL][dimM]:
//
//     ^ k-axis
//    /
//   *---------*
//  /.        /|
// *---------*-+---> m-axis
// | .       | |
// | .       | |
// | *.......|.*
// |.        |/
// *---------*
// |
// |
// \/ l-axis
//
// Along the k-axis, dimJ=12 faces form a "sphere" and
// we have in total dimI sphere in the buffers

template<typename VAL_T>
__global__ void HEALPixPadBck_bulk_k(const int padSize,
				     const int dimI,
				     const int dimJ,
				     const int dimK,
				     const int dimL,
				     const int dimM,
				     const VAL_T *__restrict__ vin,
				     VAL_T *__restrict__ vout) {

  const long long tid = ((long long)blockIdx.x)*blockDim.x + threadIdx.x;

  if (tid >= ((long long)dimI)*dimJ*dimK*dimL*dimM) {
    return;
  }
  
  const long long sliceId = tid / (dimM*dimL);

  const int i = (tid % (dimM*dimL)) / dimM;
  const int j = (tid % (dimM*dimL)) % dimM;

  const int dimLI = dimL + 2*padSize;
  const int dimMI = dimM + 2*padSize;

  vout[sliceId*dimM*dimL + i*dimM + j] = vin[sliceId*dimLI*dimMI + (padSize+i)*dimMI + padSize+j];

  return;
}

// faces functions

template<typename VAL_T>
__device__ VAL_T getTopFaceElem_d(const int k,
				  const int m,
				  const int dimL,
				  const int dimM,
				  const VAL_T *__restrict__ facePtr) {

  return facePtr[k*dimL*dimM + m];
}

template<typename VAL_T>
__device__ VAL_T getBottomFaceElem_d(const int k,
				     const int m,
				     const int dimL,
				     const int dimM,
				     const VAL_T *__restrict__ facePtr) {

  return facePtr[k*dimL*dimM + (dimL-1)*dimM + m];
}

template<typename VAL_T>
__device__ VAL_T getLeftFaceElem_d(const int k,
				   const int l,
				   const int dimL,
				   const int dimM,
				   const VAL_T *__restrict__ facePtr) {

  return facePtr[k*dimL*dimM + l*dimM];
}

template<typename VAL_T>
__device__ VAL_T getRightFaceElem_d(const int k,
				    const int l,
				    const int dimL,
				    const int dimM,
				    const VAL_T *__restrict__ facePtr) {

  return facePtr[k*dimL*dimM + l*dimM + dimM-1];
}

template<typename VAL_T>
__device__ VAL_T getT_d(const int padSize,
			const int k,
			const int p,
			const int m,
			const int dimL,
			const int dimM,
			const int faceId,
			const int faceLen,
			const VAL_T *__restrict__ sphrPtr) {

  VAL_T ret = VAL_T(0);

  switch(faceId) {
    // north faces
  case  0: ret = getLeftFaceElem_d(k, m, dimL, dimM, sphrPtr + 1*faceLen + padSize*dimM + p); break;
  case  1: ret = getLeftFaceElem_d(k, m, dimL, dimM, sphrPtr + 2*faceLen + padSize*dimM + p); break;
  case  2: ret = getLeftFaceElem_d(k, m, dimL, dimM, sphrPtr + 3*faceLen + padSize*dimM + p); break;
  case  3: ret = getLeftFaceElem_d(k, m, dimL, dimM, sphrPtr + 0*faceLen + padSize*dimM + p); break;
    // center faces
  case  4: ret = getBottomFaceElem_d(k, m, dimL, dimM, sphrPtr + 0*faceLen + padSize - p*dimM); break;
  case  5: ret = getBottomFaceElem_d(k, m, dimL, dimM, sphrPtr + 1*faceLen + padSize - p*dimM); break;
  case  6: ret = getBottomFaceElem_d(k, m, dimL, dimM, sphrPtr + 2*faceLen + padSize - p*dimM); break;
  case  7: ret = getBottomFaceElem_d(k, m, dimL, dimM, sphrPtr + 3*faceLen + padSize - p*dimM); break;
    // south faces
  case  8: ret = getBottomFaceElem_d(k, m, dimL, dimM, sphrPtr + 5*faceLen + padSize - p*dimM); break;
  case  9: ret = getBottomFaceElem_d(k, m, dimL, dimM, sphrPtr + 6*faceLen + padSize - p*dimM); break;
  case 10: ret = getBottomFaceElem_d(k, m, dimL, dimM, sphrPtr + 7*faceLen + padSize - p*dimM); break;
  case 11: ret = getBottomFaceElem_d(k, m, dimL, dimM, sphrPtr + 4*faceLen + padSize - p*dimM); break;
  }
  return ret;
}

template<typename VAL_T>
__device__ VAL_T getB_d(const int padSize,
			const int k,
			const int p,
			const int m,
			const int dimL,
			const int dimM,
			const int faceId,
			const int faceLen,
			const VAL_T *__restrict__ sphrPtr) {

  VAL_T ret = VAL_T(0);

  switch(faceId) {
    // north faces
  case  0: ret = getTopFaceElem_d(k, m, dimL, dimM, sphrPtr + 4*faceLen + padSize + p*dimM); break;
  case  1: ret = getTopFaceElem_d(k, m, dimL, dimM, sphrPtr + 5*faceLen + padSize + p*dimM); break;
  case  2: ret = getTopFaceElem_d(k, m, dimL, dimM, sphrPtr + 6*faceLen + padSize + p*dimM); break;
  case  3: ret = getTopFaceElem_d(k, m, dimL, dimM, sphrPtr + 7*faceLen + padSize + p*dimM); break;
    // center faces
  case  4: ret = getTopFaceElem_d(k, m, dimL, dimM, sphrPtr + 11*faceLen + padSize + p*dimM); break;
  case  5: ret = getTopFaceElem_d(k, m, dimL, dimM, sphrPtr +  8*faceLen + padSize + p*dimM); break;
  case  6: ret = getTopFaceElem_d(k, m, dimL, dimM, sphrPtr +  9*faceLen + padSize + p*dimM); break;
  case  7: ret = getTopFaceElem_d(k, m, dimL, dimM, sphrPtr + 10*faceLen + padSize + p*dimM); break;
    // south faces
  case  8: ret = getRightFaceElem_d(k, m, dimL, dimM, sphrPtr + 11*faceLen + padSize*dimM - p); break;
  case  9: ret = getRightFaceElem_d(k, m, dimL, dimM, sphrPtr +  8*faceLen + padSize*dimM - p); break;
  case 10: ret = getRightFaceElem_d(k, m, dimL, dimM, sphrPtr +  9*faceLen + padSize*dimM - p); break;
  case 11: ret = getRightFaceElem_d(k, m, dimL, dimM, sphrPtr + 10*faceLen + padSize*dimM - p); break;
  }
  return ret;
}

template<typename VAL_T>
__device__ VAL_T getL_d(const int padSize,
			const int k,
			const int p,
			const int m,
			const int dimL,
			const int dimM,
			const int faceId,
			const int faceLen,
			const VAL_T *__restrict__ sphrPtr) {

  VAL_T ret = VAL_T(0);

  switch(faceId) {
    // north faces
  case  0: ret = getTopFaceElem_d(k, m, dimL, dimM, sphrPtr + 3*faceLen + padSize + p*dimM); break;
  case  1: ret = getTopFaceElem_d(k, m, dimL, dimM, sphrPtr + 0*faceLen + padSize + p*dimM); break;
  case  2: ret = getTopFaceElem_d(k, m, dimL, dimM, sphrPtr + 1*faceLen + padSize + p*dimM); break;
  case  3: ret = getTopFaceElem_d(k, m, dimL, dimM, sphrPtr + 2*faceLen + padSize + p*dimM); break;
    // center faces
  case  4: ret = getRightFaceElem_d(k, m, dimL, dimM, sphrPtr + 3*faceLen + padSize*dimM - p); break;
  case  5: ret = getRightFaceElem_d(k, m, dimL, dimM, sphrPtr + 0*faceLen + padSize*dimM - p); break;
  case  6: ret = getRightFaceElem_d(k, m, dimL, dimM, sphrPtr + 1*faceLen + padSize*dimM - p); break;
  case  7: ret = getRightFaceElem_d(k, m, dimL, dimM, sphrPtr + 2*faceLen + padSize*dimM - p); break;
    // south faces
  case  8: ret = getRightFaceElem_d(k, m, dimL, dimM, sphrPtr + 4*faceLen + padSize*dimM - p); break;
  case  9: ret = getRightFaceElem_d(k, m, dimL, dimM, sphrPtr + 5*faceLen + padSize*dimM - p); break;
  case 10: ret = getRightFaceElem_d(k, m, dimL, dimM, sphrPtr + 6*faceLen + padSize*dimM - p); break;
  case 11: ret = getRightFaceElem_d(k, m, dimL, dimM, sphrPtr + 7*faceLen + padSize*dimM - p); break;
  }
  return ret;
}

template<typename VAL_T>
__device__ VAL_T getR_d(const int padSize,
			const int k,
			const int p,
			const int m,
			const int dimL,
			const int dimM,
			const int faceId,
			const int faceLen,
			const VAL_T *__restrict__ sphrPtr) {

  VAL_T ret = VAL_T(0);

  switch(faceId) {
    // north faces
  case  0: ret = getLeftFaceElem_d(k, m, dimL, dimM, sphrPtr + 5*faceLen + padSize*dimM + p); break;
  case  1: ret = getLeftFaceElem_d(k, m, dimL, dimM, sphrPtr + 6*faceLen + padSize*dimM + p); break;
  case  2: ret = getLeftFaceElem_d(k, m, dimL, dimM, sphrPtr + 7*faceLen + padSize*dimM + p); break;
  case  3: ret = getLeftFaceElem_d(k, m, dimL, dimM, sphrPtr + 4*faceLen + padSize*dimM + p); break;
    // center faces
  case  4: ret = getLeftFaceElem_d(k, m, dimL, dimM, sphrPtr +  8*faceLen + padSize*dimM + p); break;
  case  5: ret = getLeftFaceElem_d(k, m, dimL, dimM, sphrPtr +  9*faceLen + padSize*dimM + p); break;
  case  6: ret = getLeftFaceElem_d(k, m, dimL, dimM, sphrPtr + 10*faceLen + padSize*dimM + p); break;
  case  7: ret = getLeftFaceElem_d(k, m, dimL, dimM, sphrPtr + 11*faceLen + padSize*dimM + p); break;
    // south faces
  case  8: ret = getBottomFaceElem_d(k, m, dimL, dimM, sphrPtr +  9*faceLen + padSize - p*dimM); break;
  case  9: ret = getBottomFaceElem_d(k, m, dimL, dimM, sphrPtr + 10*faceLen + padSize - p*dimM); break;
  case 10: ret = getBottomFaceElem_d(k, m, dimL, dimM, sphrPtr + 11*faceLen + padSize - p*dimM); break;
  case 11: ret = getBottomFaceElem_d(k, m, dimL, dimM, sphrPtr +  8*faceLen + padSize - p*dimM); break;
  }
  return ret;
}

template<typename VAL_T>
__global__ void HEALPixPadBck_haloTB_k(const int padSize,
				       const int dimI,
				       const int dimJ, // = 12
				       const int dimK,
				       const int dimL,
				       const int dimM,
				       const VAL_T *__restrict__ vin,
				       VAL_T *__restrict__ vout) {

  const long long tid = ((long long)blockIdx.x)*blockDim.x + threadIdx.x;

  if (tid >= dimI*dimJ*dimK*dimM*padSize) {
    return;
  }

  const long long sphrId = tid / (dimJ*dimK*dimM*padSize);
  const long long faceId = (tid - sphrId*(dimJ*dimK*dimM*padSize)) / (dimK*dimM*padSize);

  const int dimLI = dimL + 2*padSize;
  const int dimMI = dimM + 2*padSize;

  const long long faceLenI = ((long long)dimK)*dimLI*dimMI;
  const long long faceLenO = ((long long)dimK)*dimL *dimM;

  const VAL_T *__restrict__ sphrPtrI = vin  +  sphrId*dimJ          *faceLenI;
  VAL_T *__restrict__ facePtrO = vout + (sphrId*dimJ + faceId)*faceLenO;

  const int k = (tid / (padSize*dimM)) % dimK;
  const int p = (tid /          dimM)  % padSize;
  const int m =  tid                   % dimM;

  // copy top    face
  // copy bottom face
  facePtrO[k*dimL*dimM +    (padSize-1)*dimM + m - p*dimM] += getT_d(padSize, k, p, m, dimLI, dimMI, faceId, faceLenI, sphrPtrI);
  facePtrO[k*dimL*dimM + (dimL-padSize)*dimM + m + p*dimM] += getB_d(padSize, k, p, m, dimLI, dimMI, faceId, faceLenI, sphrPtrI);

  return;
}

template<typename VAL_T>
__global__ void HEALPixPadBck_haloLR_k(const int padSize,
				       const int dimI,
				       const int dimJ, // = 12
				       const int dimK,
				       const int dimL,
				       const int dimM,
				       const VAL_T *__restrict__ vin,
				       VAL_T *__restrict__ vout) {

  const long long tid = ((long long)blockIdx.x)*blockDim.x + threadIdx.x;

  if (tid >= ((long long)dimI)*dimJ*dimK*dimM*padSize) {
    return;
  }

  const long long sphrId = tid / (dimJ*dimK*dimM*padSize);
  const long long faceId = (tid - sphrId*(dimJ*dimK*dimM*padSize)) / (dimK*dimM*padSize);

  const int dimLI = dimL + 2*padSize;
  const int dimMI = dimM + 2*padSize;

  const long long faceLenI = ((long long)dimK)*dimLI*dimMI;
  const long long faceLenO = ((long long)dimK)*dimL *dimM;

  const VAL_T *__restrict__ sphrPtrI = vin  +  sphrId*dimJ          *faceLenI;
  VAL_T *__restrict__ facePtrO = vout + (sphrId*dimJ + faceId)*faceLenO;

  const int k = (tid / (padSize*dimM)) % dimK;
  const int p = (tid /          dimM)  % padSize;
  const int m =  tid                   % dimM;

  // copy left   face
  // copy right  face
  facePtrO[k*dimL*dimM + m*dimM +    padSize-1 - p] += getL_d(padSize, k, p, m, dimLI, dimMI, faceId, faceLenI, sphrPtrI);
  facePtrO[k*dimL*dimM + m*dimM + dimM-padSize + p] += getR_d(padSize, k, p, m, dimLI, dimMI, faceId, faceLenI, sphrPtrI);

  return;
}

// corners functions

template<typename VAL_T>
__device__ VAL_T getTopLeftCornerElem_d(const int k,
					const int dimL,
					const int dimM,
					const VAL_T *__restrict__ facePtr) {

  return facePtr[k*dimL*dimM];
}

template<typename VAL_T>
__device__ VAL_T getTopRightCornerElem_d(const int k,
					 const int dimL,
					 const int dimM,
					 const VAL_T *__restrict__ facePtr) {

  return facePtr[k*dimL*dimM + dimM-1];
}

template<typename VAL_T>
__device__ VAL_T getBottomLeftCornerElem_d(const int k,
					   const int dimL,
					   const int dimM,
					   const VAL_T *__restrict__ facePtr) {

  return facePtr[k*dimL*dimM + (dimL-1)*dimM];
}

template<typename VAL_T>
__device__ VAL_T getBottomRightCornerElem_d(const int k,
					    const int dimL,
					    const int dimM,
					    const VAL_T *__restrict__ facePtr) {

  return facePtr[k*dimL*dimM + dimL*dimM-1];
}

template<typename VAL_T>
__device__ VAL_T getTL_d(const int padSize,
			 const int p,
			 const int q,
			 const int k,
			 const int dimL,
			 const int dimM,
			 const int faceId,
			 const int faceLen,
			 const VAL_T *__restrict__ sphrPtr) {

  VAL_T ret = VAL_T(0);

  const int pinv = padSize-1 - p;
  const int qinv = padSize-1 - q;

  // offset from neighbor's corner
  // for non equatorial faces
  const int padOff = pinv*dimM + qinv;

  switch(faceId) {
    // north faces
  case  0: ret = getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 2*faceLen + padOff); break;
  case  1: ret = getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 3*faceLen + padOff); break;
  case  2: ret = getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 0*faceLen + padOff); break;
  case  3: ret = getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 1*faceLen + padOff); break;
    // center faces
  case  4:
  case  5:
  case  6:
  case  7: break;
    // south faces
  case  8: ret = getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr + 0*faceLen - padOff); break;
  case  9: ret = getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr + 1*faceLen - padOff); break;
  case 10: ret = getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr + 2*faceLen - padOff); break;
  case 11: ret = getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr + 3*faceLen - padOff); break;
  }
  return ret;
}

template<typename VAL_T>
__device__ VAL_T getTR_d(const int padSize,
			 const int p,
			 const int q,
			 const int k,
			 const int dimL,
			 const int dimM,
			 const int faceId,
			 const int faceLen,
			 const VAL_T *__restrict__ sphrPtr) {

  VAL_T ret = VAL_T(0);

  const int pinv = padSize-1 - p;

  // offset from neighbor's corner
  // for non equatorial faces
  const int padOff = -pinv*dimM + q;

  switch(faceId) {
    // north faces
  case  0: ret = getBottomLeftCornerElem_d(k, dimL, dimM, sphrPtr + 1*faceLen + padOff); break;
  case  1: ret = getBottomLeftCornerElem_d(k, dimL, dimM, sphrPtr + 2*faceLen + padOff); break;
  case  2: ret = getBottomLeftCornerElem_d(k, dimL, dimM, sphrPtr + 3*faceLen + padOff); break;
  case  3: ret = getBottomLeftCornerElem_d(k, dimL, dimM, sphrPtr + 0*faceLen + padOff); break;
    // center faces
  case  4: ret = getBottomLeftCornerElem_d(k, dimL, dimM, sphrPtr + 5*faceLen + padOff); break;
  case  5: ret = getBottomLeftCornerElem_d(k, dimL, dimM, sphrPtr + 6*faceLen + padOff); break;
  case  6: ret = getBottomLeftCornerElem_d(k, dimL, dimM, sphrPtr + 7*faceLen + padOff); break;
  case  7: ret = getBottomLeftCornerElem_d(k, dimL, dimM, sphrPtr + 4*faceLen + padOff); break;
    // south faces
  case  8: ret = getBottomLeftCornerElem_d(k, dimL, dimM, sphrPtr +  9*faceLen + padOff); break;
  case  9: ret = getBottomLeftCornerElem_d(k, dimL, dimM, sphrPtr + 10*faceLen + padOff); break;
  case 10: ret = getBottomLeftCornerElem_d(k, dimL, dimM, sphrPtr + 11*faceLen + padOff); break;
  case 11: ret = getBottomLeftCornerElem_d(k, dimL, dimM, sphrPtr +  8*faceLen + padOff); break;
  }

  // offsets from neighbor' corners
  // for north and south faces
  const int topLeftPadOff  = (q+1 + p)*dimM + q;

  // north faces get values in the top left tringle
  // of their corner square from half the corner of
  // other faces

  if (p+q < padSize-1) {
    switch(faceId) {
      // north faces
    case  0: ret += getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 5*faceLen + topLeftPadOff); break;
    case  1: ret += getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 6*faceLen + topLeftPadOff); break;
    case  2: ret += getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 7*faceLen + topLeftPadOff); break;
    case  3: ret += getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 4*faceLen + topLeftPadOff); break;
    }
  }

  if (p == 0) {
    switch(faceId) {
      // north faces
    case  0: ret += getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 5*faceLen + q*dimM + q) / VAL_T(2); break;
    case  1: ret += getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 6*faceLen + q*dimM + q) / VAL_T(2); break;
    case  2: ret += getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 7*faceLen + q*dimM + q) / VAL_T(2); break;
    case  3: ret += getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 4*faceLen + q*dimM + q) / VAL_T(2); break;
    }
  }

  const int qinv = padSize-1 - q;
  const int bottomRightOff = -pinv*dimM - (pinv+1 + qinv);

  // south faces get values in the bottom right tringle
  // of their corner square from half the corner of
  // other faces

  if (p+q > padSize-1) {
    switch(faceId) {
      // south faces
    case  8: ret += getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr +  5*faceLen + bottomRightOff); break;
    case  9: ret += getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr +  6*faceLen + bottomRightOff); break;
    case 10: ret += getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr +  7*faceLen + bottomRightOff); break;
    case 11: ret += getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr +  4*faceLen + bottomRightOff); break;
    }
  }

  if (q == padSize-1) {
    switch(faceId) {
      // south faces
    case  8: ret += getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr +  5*faceLen - pinv*dimM - pinv) / VAL_T(2); break;
    case  9: ret += getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr +  6*faceLen - pinv*dimM - pinv) / VAL_T(2); break;
    case 10: ret += getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr +  7*faceLen - pinv*dimM - pinv) / VAL_T(2); break;
    case 11: ret += getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr +  4*faceLen - pinv*dimM - pinv) / VAL_T(2); break;
    }
  }

  return ret;
}

template<typename VAL_T>
__device__ VAL_T getBL_d(const int padSize,
			 const int p,
			 const int q,
			 const int k,
			 const int dimL,
			 const int dimM,
			 const int faceId,
			 const int faceLen,
			 const VAL_T *__restrict__ sphrPtr) {

  VAL_T ret = VAL_T(0);

  const int qinv = padSize-1 - q;

  // offset from neighbor's corner
  // for non equatorial faces
  const int padOff = p*dimM - qinv;

  switch(faceId) {
    // north faces
  case  0: ret = getTopRightCornerElem_d(k, dimL, dimM, sphrPtr + 3*faceLen + padOff); break;
  case  1: ret = getTopRightCornerElem_d(k, dimL, dimM, sphrPtr + 0*faceLen + padOff); break;
  case  2: ret = getTopRightCornerElem_d(k, dimL, dimM, sphrPtr + 1*faceLen + padOff); break;
  case  3: ret = getTopRightCornerElem_d(k, dimL, dimM, sphrPtr + 2*faceLen + padOff); break;
    // center faces
  case  4: ret = getTopRightCornerElem_d(k, dimL, dimM, sphrPtr + 7*faceLen + padOff); break;
  case  5: ret = getTopRightCornerElem_d(k, dimL, dimM, sphrPtr + 4*faceLen + padOff); break;
  case  6: ret = getTopRightCornerElem_d(k, dimL, dimM, sphrPtr + 5*faceLen + padOff); break;
  case  7: ret = getTopRightCornerElem_d(k, dimL, dimM, sphrPtr + 6*faceLen + padOff); break;
    // south faces
  case  8: ret = getTopRightCornerElem_d(k, dimL, dimM, sphrPtr + 11*faceLen + padOff); break;
  case  9: ret = getTopRightCornerElem_d(k, dimL, dimM, sphrPtr +  8*faceLen + padOff); break;
  case 10: ret = getTopRightCornerElem_d(k, dimL, dimM, sphrPtr +  9*faceLen + padOff); break;
  case 11: ret = getTopRightCornerElem_d(k, dimL, dimM, sphrPtr + 10*faceLen + padOff); break;
  }

  // offsets from neighbor' corners
  // for north and south faces
  const int bottomLeftPadOff  = p*dimM + p+1+q; //(q+1 + p)*dimM + q;

  if (p+q < padSize-1) {
    switch(faceId) {
      // north faces
    case  0: ret += getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 4*faceLen + bottomLeftPadOff); break;
    case  1: ret += getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 5*faceLen + bottomLeftPadOff); break;
    case  2: ret += getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 6*faceLen + bottomLeftPadOff); break;
    case  3: ret += getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 7*faceLen + bottomLeftPadOff); break;
    }
  }

  if (q == 0) {
    switch(faceId) {
      // north faces
    case  0: ret += getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 4*faceLen + p*dimM + p) / VAL_T(2); break;
    case  1: ret += getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 5*faceLen + p*dimM + p) / VAL_T(2); break;
    case  2: ret += getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 6*faceLen + p*dimM + p) / VAL_T(2); break;
    case  3: ret += getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 7*faceLen + p*dimM + p) / VAL_T(2); break;
    }
  }

  //const int pinv = padSize-1 - p;
  const int bottomRightOff = (2*padSize - p - q -1)*dimM + qinv; //-(pinv + q-1)*dimM + q;

  // south faces get values in the bottom right tringle
  // of their corner square from half the corner of
  // other faces
  if (p+q > padSize-1) {
    switch(faceId) {
      // south faces
    case  8: ret += getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr +  4*faceLen - bottomRightOff); break;
    case  9: ret += getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr +  5*faceLen - bottomRightOff); break;
    case 10: ret += getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr +  6*faceLen - bottomRightOff); break;
    case 11: ret += getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr +  7*faceLen - bottomRightOff); break;
    }
  }

  if (p == padSize-1) {
    switch(faceId) {
      // south faces
    case  8: ret += getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr +  4*faceLen - qinv*dimM - qinv) / VAL_T(2); break;
    case  9: ret += getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr +  5*faceLen - qinv*dimM - qinv) / VAL_T(2); break;
    case 10: ret += getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr +  6*faceLen - qinv*dimM - qinv) / VAL_T(2); break;
    case 11: ret += getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr +  7*faceLen - qinv*dimM - qinv) / VAL_T(2); break;
    }
  }

  return ret;
}

template<typename VAL_T>
__device__ VAL_T getBR_d(const int padSize,
			 const int p,
			 const int q,
			 const int k,
			 const int dimL,
			 const int dimM,
			 const int faceId,
			 const int faceLen,
			 const VAL_T *__restrict__ sphrPtr) {

  VAL_T ret = VAL_T(0);

  // offset from neighbor's corner
  // for non equatorial faces
  const int padOff = p*dimM + q;

  switch(faceId) {
    // north faces
  case  0: ret = getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr +  8*faceLen + padOff); break;
  case  1: ret = getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr +  9*faceLen + padOff); break;
  case  2: ret = getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 10*faceLen + padOff); break;
  case  3: ret = getTopLeftCornerElem_d(k, dimL, dimM, sphrPtr + 11*faceLen + padOff); break;
    // center faces
  case  4:
  case  5:
  case  6:
  case  7: break;
    // south faces
  case  8: ret = getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr + 10*faceLen - padOff); break;
  case  9: ret = getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr + 11*faceLen - padOff); break;
  case 10: ret = getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr +  8*faceLen - padOff); break;
  case 11: ret = getBottomRightCornerElem_d(k, dimL, dimM, sphrPtr +  9*faceLen - padOff); break;
  }
  return ret;
}

template<typename VAL_T>
__global__ void HEALPixPadBck_haloCR_k(const int padSize,
				       const int dimI,
				       const int dimJ, // = 12
				       const int dimK,
				       const int dimL,
				       const int dimM,
				       const VAL_T *__restrict__ vin,
				       VAL_T *__restrict__ vout) {

  const long long tid = ((long long)blockIdx.x)*blockDim.x + threadIdx.x;

  if (tid >= ((long long)dimI)*dimJ*dimK*padSize*padSize) {
    return;
  }

  const long long sphrId = tid / (dimJ*dimK*padSize*padSize);
  const long long faceId = (tid - sphrId*(dimJ*dimK*padSize*padSize)) / (dimK*padSize*padSize);

  const int dimLI = dimL + 2*padSize;
  const int dimMI = dimM + 2*padSize;

  const long long faceLenI = ((long long)dimK)*dimLI*dimMI;
  const long long faceLenO = ((long long)dimK)*dimL *dimM;

  const VAL_T *__restrict__ sphrPtrI = vin  +  sphrId*dimJ          *faceLenI;
  VAL_T *__restrict__ facePtrO = vout + (sphrId*dimJ + faceId)*faceLenO;

  const int k = (tid /  (padSize*padSize)) % dimK;
  const int p = (tid /           padSize)  % padSize;
  const int q =  tid                       % padSize;

  // copy top-left     corner
  // copy top-right    corner
  // copy bottom-left  corner
  // copy bottom-right corner
  facePtrO[k*dimL*dimM                                      + p*dimM + q] += getTL_d(padSize, p, q, k, dimLI, dimMI, faceId, faceLenI, sphrPtrI);
  facePtrO[k*dimL*dimM                       + dimM-padSize + p*dimM + q] += getTR_d(padSize, p, q, k, dimLI, dimMI, faceId, faceLenI, sphrPtrI);
  facePtrO[k*dimL*dimM + (dimL-padSize)*dimM                + p*dimM + q] += getBL_d(padSize, p, q, k, dimLI, dimMI, faceId, faceLenI, sphrPtrI);
  facePtrO[k*dimL*dimM + (dimL-padSize)*dimM + dimM-padSize + p*dimM + q] += getBR_d(padSize, p, q, k, dimLI, dimMI, faceId, faceLenI, sphrPtrI);

  return;
}

template<typename REAL_T>
void HEALPixPadBck(int padSize,
		   int dimI, // batch size
		   int dimJ, // 12
		   int dimK, // no. of channels
		   int dimL, // face no. of rows of dataOut_d (dataIn_d has dimL+2 rows)
		   int dimM, // face no. of cols of dataOut_d (dataIn_d has dimM+2 cols)
		   REAL_T *dataIn_d,
		   REAL_T *dataOut_d,
		   hipStream_t stream) {

  if (dimI*dimJ*dimK*dimL*dimM <= 0) {
    fprintf(stderr, "%s:%d: error, one or more dimension is less than or equal zero!\n", __func__, __LINE__);
    exit(EXIT_FAILURE);
  }

  if (dimJ != 12) {
    fprintf(stderr, "%s:%d: error, dimJ must be equal to 12!\n", __func__, __LINE__);
    exit(EXIT_FAILURE);
  }

  if (dimL != dimM) {
    fprintf(stderr, "%s:%d: error, dimL must be equal to dimM!\n", __func__, __LINE__);
    exit(EXIT_FAILURE);
  }

  if (padSize > dimL) {
    fprintf(stderr, "%s:%d: error, padSize and less than or equal dimL (or dimM)\n", __func__, __LINE__);
    exit(EXIT_FAILURE);
  }

  // copy bulk
  const int nth_b = THREADS;
  const int nbl_b = DIV_UP(dimI*dimJ*dimK*dimL*dimM, nth_b);

  HEALPixPadBck_bulk_k<<<nbl_b, nth_b, 0, stream>>>(padSize, dimI, dimJ, dimK, dimL, dimM, dataIn_d, dataOut_d);

  CHECK_ERROR("HEALPixPad_bck_bulk_k");

  // copy haloes
  const int nth_f = THREADS;
  const int nbl_f = DIV_UP(dimI*dimJ*dimK*dimM*padSize, nth_f);

  HEALPixPadBck_haloTB_k<<<nbl_f, nth_f, 0, stream>>>(padSize, dimI, dimJ, dimK, dimL, dimM, dataIn_d, dataOut_d);
  CHECK_ERROR("HEALPixPadBck_haloTB_k");
  HEALPixPadBck_haloLR_k<<<nbl_f, nth_f, 0, stream>>>(padSize, dimI, dimJ, dimK, dimL, dimM, dataIn_d, dataOut_d);
  CHECK_ERROR("HEALPixPadBck_haloLR_k");

  const int nth_c = THREADS;
  const int nbl_c = DIV_UP(dimI*dimJ*dimK*padSize*padSize, nth_c);

  HEALPixPadBck_haloCR_k<<<nbl_c, nth_c, 0, stream>>>(padSize, dimI, dimJ, dimK, dimL, dimM, dataIn_d, dataOut_d);

  CHECK_ERROR("HEALPixPadBck_haloCR_k");

  //CHECK_CUDA(hipStreamSynchronize(stream));

  return;
}

void HEALPixPad_bwd_fp32(int padSize,
			 int dimI,
			 int dimJ,
			 int dimK,
			 int dimL,
			 int dimM,
			 float *dataIn_d,
			 float *dataOut_d,
			 hipStream_t stream) {
  
  HEALPixPadBck<float>(padSize, dimI, dimJ, dimK, dimL, dimM, dataIn_d, dataOut_d, stream);
  
  return;
}

void HEALPixPad_bwd_fp64(int padSize,
			 int dimI,
			 int dimJ,
			 int dimK,
			 int dimL,
			 int dimM,
			 double *dataIn_d,
			 double *dataOut_d,
			 hipStream_t stream) {
  
  HEALPixPadBck<double>(padSize, dimI, dimJ, dimK, dimL, dimM, dataIn_d, dataOut_d, stream);
  
  return;
}

std::vector<torch::Tensor> healpixpad_cuda_backward(
						    torch::Tensor ginput,
						    int pad) {
  const auto batch_size = ginput.size(0);
  const auto num_faces = ginput.size(1);
  const auto num_channels = ginput.size(2);
  // the face size is the size of the output gradient
  const auto face_size = ginput.size(3) - 2*pad;
  int64_t shape[5] = {batch_size, num_faces, num_channels, face_size, face_size};

  // allocate output tensor
  c10::TensorOptions options = c10::TensorOptions().device(ginput.device()).dtype(ginput.dtype());
  torch::IntArrayRef size = c10::makeArrayRef<int64_t>(shape, 5);
  auto goutput = torch::empty(size, options);

  // get cuda stream:
  hipStream_t my_stream = c10::cuda::getCurrentCUDAStream(ginput.device().index()).stream();

  switch (ginput.scalar_type()) {
  case torch::ScalarType::Double:
    HEALPixPadBck<double>(pad,
			  batch_size,
			  num_faces,
			  num_channels,
			  face_size,
			  face_size,
			  ginput.data_ptr<double>(),
			  goutput.data_ptr<double>(),
			  my_stream);
    break;
  case torch::ScalarType::Float:
    HEALPixPadBck<float>(pad,
			 batch_size,
			 num_faces,
			 num_channels,
			 face_size,
			 face_size,
			 ginput.data_ptr<float>(),
			 goutput.data_ptr<float>(),
			 my_stream);
    break;
  case torch::ScalarType::Half:
    HEALPixPadBck<at::Half>(pad,
			    batch_size,
			    num_faces,
			    num_channels,
			    face_size,
			    face_size,
			    ginput.data_ptr<at::Half>(),
			    goutput.data_ptr<at::Half>(),
			    my_stream);
    break;
  case torch::ScalarType::BFloat16:
    HEALPixPadBck<at::BFloat16>(pad,
			    batch_size,
			    num_faces,
			    num_channels,
			    face_size,
			    face_size,
			    ginput.data_ptr<at::BFloat16>(),
			    goutput.data_ptr<at::BFloat16>(),
			    my_stream);
    break;
  }

  return {goutput};
}  
